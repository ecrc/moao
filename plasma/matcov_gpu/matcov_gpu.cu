#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<hipblas.h>
#include<stdio.h>
#include "matcov.h"
#include "matcov_gpu.h"

/*  Tuning parameters of tbulateDPHI kernel*/
#define tabDPHI_thread_x	(256)

/*	Tuning parameters of matcov GPU Kernel */
// Thread block size (x, y), 
// max #threads per block is 512 for fermi and 1024 for kepler
#define matcov_thread_x	(8)
#define matcov_thread_y	(8)

#define CUDA_ERROR_CHECK
 
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )
 
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif
 
    return;
}
 
inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
 
    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif
 
    return;
}

//============================================================================================
//================================= AUX FUNCTIONS ============================================
//============================================================================================
void process_error(hipError_t e, const char* str)
{
	if(e != hipSuccess)
	{
		printf("*** Error %s: %s \n", str, hipGetErrorString(e));
		exit(1);
	}
}
//-----------------------------------------------------------------------
double* arr2dAlloc_gpu(long nbLin, long nbCol)
/* DOCUMENT  array = arr2dAlloc(nblin,nbcol)

 Allocates a 2d array (double).
 */
{
	hipError_t e;
	double* tableau;
	e = hipMalloc((void**)&tableau, sizeof(double) * nbCol * nbLin);
	process_error(e, "gpu alloc tableau2");
	return tableau;
}

void arr2dFree_gpu(double *tableau)
/* DOCUMENT  arr2dFree(array)

 Free a 2d array (double).
 */
{
	if(tableau)hipFree(tableau);
}

//============================================================================================
//============================= tabDPHI KERNEL(s) ============================================
//============================================================================================
__device__ double macdo_x56_gpu(double x, int k)
/* DOCUMENT  macdo_x56_gpu(x)

 Computation of the function
 f(x) = x^(5/6)*K_{5/6}(x)
 using a series for the esimation of K_{5/6}, taken from Rod Conan thesis :
 K_a(x)=1/2 \sum_{n=0}^\infty \frac{(-1)^n}{n!}
 \left(\Gamma(-n-a) (x/2)^{2n+a} + \Gamma(-n+a) (x/2)^{2n-a} \right) ,
 with a = 5/6.

 Setting x22 = (x/2)^2, setting uda = (1/2)^a, and multiplying by x^a,
 this becomes :
 x^a * Ka(x) = 0.5 $ -1^n / n! [ G(-n-a).uda x22^(n+a) + G(-n+a)/uda x22^n ]
 Then we use the following recurrence formulae on the following quantities :
 G(-(n+1)-a) = G(-n-a) / -a-n-1
 G(-(n+1)+a) = G(-n+a) /  a-n-1
 (n+1)! = n! * (n+1)
 x22^(n+1) = x22^n * x22
 and at each iteration on n, one will use the values already computed at step (n-1).
 The values of G(a) and G(-a) are hardcoded instead of being computed.

 The first term of the series has also been skipped, as it
 vanishes with another term in the expression of Dphi.

 SEE ALSO:
 */
{
	const double a = 5. / 6.;
	const double x2a = pow(x, 2. * a), x22 = x * x / 4.;
	double x2n;               // x^2.a, etc
	double s = 0.0;
	int n;
	
	const double Ga[11] = { 0, 12.067619015983075, 5.17183672113560444,
    						0.795667187867016068, 0.0628158306210802181, 0.00301515986981185091,
    						9.72632216068338833e-05, 2.25320204494595251e-06, 3.93000356676612095e-08,
    						5.34694362825451923e-10, 5.83302941264329804e-12 };

	const double Gma[11] = { -3.74878707653729304, -2.04479295083852408,
    						-0.360845814853857083, -0.0313778969438136685, -0.001622994669507603,
    						-5.56455315259749673e-05, -1.35720808599938951e-06,
   							-2.47515152461894642e-08, -3.50257291219662472e-10,
    						-3.95770950530691961e-12, -3.65327031259100284e-14 };

	x2n = 0.5;                           // init (1/2) * x^0
	
	s = Gma[0] * x2a;
	s *= x2n;
	
	// prepare recurrence iteration for next step
	x2n *= x22;    // x^n

	#pragma unroll
	for (n = 1; n <= 10; n++) 
	{
  		s += (Gma[n] * x2a + Ga[n]) * x2n;
  		// prepare recurrence iteration for next step
  		x2n *= x22;    // x^n
	}
	return s;
}
//------------------------------------------------------------------------------------
__device__ double asymp_macdo_gpu(double x)
/* DOCUMENT asymp_macdo_gpu(x)

 Computes a term involved in the computation of the phase struct
 function with a finite outer scale according to the Von-Karman
 model. The term involves the MacDonald function (modified bessel
 function of second kind) K_{5/6}(x), and the algorithm uses the
 asymptotic form for x ~ infinity.
 Warnings :
 - This function makes a doubleing point interrupt for x=0
 and should not be used in this case.
 - Works only for x>0.

 SEE ALSO:
 */
{
	// k2 is the value for
	// gamma_R(5./6)*2^(-1./6)
	const double k2 = 1.00563491799858928388289314170833;
	const double k3 = 1.25331413731550012081;   //  sqrt(pi/2)
	const double a1 = 0.22222222222222222222;   //  2/9
	const double a2 = -0.08641975308641974829;  //  -7/89
	const double a3 = 0.08001828989483310284;   // 175/2187
	double res;
	double x_1;
	
	x_1 = 1. / x;
	res = k2
	      - k3 * exp(-x) * pow(x, 1 / 3.)
    	  * (1.0 + x_1 * (a1 + x_1 * (a2 + x_1 * a3)));
	return res;
}
//------------------------------------------------------------------------------------
__device__ double rodconan_gpu(double r, double L0, int k)
/* DOCUMENT rodconan_gpu(r,L0,k=)
 The phase structure function is computed from the expression
 Dphi(r) = k1  * L0^(5./3) * (k2 - (2.pi.r/L0)^5/6 K_{5/6}(2.pi.r/L0))

 For small r, the expression is computed from a development of
 K_5/6 near 0. The value of k2 is not used, as this same value
 appears in the series and cancels with k2.
 For large r, the expression is taken from an asymptotic form.

 SEE ALSO:
 */
{
	const double pi = 3.1415926535897932384626433;
	double res = 0;
	
	// k1 is the value of :
	// 2*gamma_R(11./6)*2^(-5./6)*pi^(-8./3)*(24*gamma_R(6./5)/5.)^(5./6);
	const double k1 = 0.1716613621245709486;
	const double dprf0 = (2 * pi / L0) * r;
	// k2 is the value for gamma_R(5./6)*2^(-1./6),
	// but is now unused
	// k2 = 1.0056349179985892838;
	
	// Xlim = 0.75*2*pi;   // = 4.71239
	if (dprf0 > 4.71239)
		res = asymp_macdo_gpu(dprf0);
	else
		res = -macdo_x56_gpu(dprf0, k);

	res *= k1 * pow(L0, 5. / 3);
	return res;
}



__global__ void tabulateDPHI_gpu_kernel(double* tabDPHI_d, double* L0diff_d, long Nl0, long Ndphi, double convert)
{
	const int tx = threadIdx.x;
	const int ty = blockIdx.x;
	
	const int tid = ty * blockDim.x + tx;
	int l = tid / Ndphi;
	int j = tid % Ndphi;
	
	if(tid >= (Nl0*Ndphi) ) return;
	
	tabDPHI_d[tid] = rodconan_gpu((double)j / convert, L0diff_d[l], 10);
	
	//double* mytabDPHI = tabDPHI_d + (l * Ndphi);
	//
	//int j, k;
	//#pragma unroll
	//for(k = 0; k < (Ndphi/tabDPHI_thread_x); k++)
	//{
	//	j = k * tabDPHI_thread_x + tx;
	//	mytabDPHI[j] = rodconan_gpu(rr_d[j], L0diff_d[l], 10);
	//}
	//
	//k = (Ndphi/tabDPHI_thread_x);
	//if(tx < (Ndphi%tabDPHI_thread_x) )
	//{
	//	j = k * tabDPHI_thread_x + tx;
	//	mytabDPHI[j] = rodconan_gpu(rr_d[j], L0diff_d[l], 10);
	//}
}
//------------------------------------------------------------------------------------
double* tabulateDPHI_gpu(struct tomo_struct tomo, long Ndphi, long *indexL0, int* Nl0_, double convert)
//void tabulateDPHI_gpu(double* tabDPHI_d, double* rr_d,struct tomo_struct tomo, long Ndphi, long *indexL0_h)
/* DOCUMENT tabDPHI = tabulateDPHI(rr,tomo,Ndphi, indexL0)
 <tomo>            :  structure with all the needed information
 <Ndphi>           :  size of rr
 <indexL0>         :  link between the index of the studied layer and the index of the precomputed one. 

 Computes the phase structure function for a separation rr(x,y).
 The r0 is not taken into account : the final result of DPHI(x,y,L0)
 has to be scaled with r0^-5/3, with r0 expressed in meters, to get
 the right value.

 Computes the phase structure for each different L0 and give a array (indexL0) to link the index of the layer i and the index of tabDPHI : for the layer l, DPHI = DPHI( du, dv, indexL0[l],rr,tabDPHI, convert).
 SEE ALSO: DPHI
 */
{
	//Search the different L0 and build indexL0
	const long Nlayer = tomo.Nlayer;
	long i, j;
	int cpt = 1;
	double tmp[Nlayer];
	hipError_t e;
	
	tmp[0] = tomo.L0[0];
	indexL0[0] = 0;

	for (i = 1; i < Nlayer; i++) 
	{
		j = 0;
		const double l0 = tomo.L0[i];
		
		while ((j < cpt) && (tmp[j] != l0)) {j++;}
		
		indexL0[i] = j;
		
		if (j == cpt) 
		{
			tmp[j] = l0;
			cpt++;
		}
	}

	const int Nl0 = cpt;
	double L0diff[Nl0];
	double* L0diff_d;
	
	// allocate space for L0
	e = hipMalloc((void**)&L0diff_d, Nl0*sizeof(double));
	process_error(e, "alloc gpu L0diff_d");
	
	for (i = 0; i < Nl0; i++) 
	{
		L0diff[i] = tmp[i];
	}
	
	// offload L0diff
	e = hipMemcpy(L0diff_d, L0diff, Nl0*sizeof(double), hipMemcpyHostToDevice);
	process_error(e, "offload L0diff");
	
	//précalcul de DPHI : que pour chaque différent L0
	double* tabDPHI_d = arr2dAlloc_gpu(Nl0, Ndphi);
	
	// gpu kernel goes here
	//for (l = 0; l < Nl0; l++) 
	//{
	//	#ifdef USE_OPENMP
	//	#pragma omp parallel num_threads(tomo.ncpu)
	//	#pragma omp for nowait
	//	#endif
	//	for (j = 0; j < Ndphi; j++) 
	//	{
	//		tabDPHI[l][j] = rodconan_gpu(rr[j], L0diff[l], 10);
	//	}
	//}
	// Assume one thread per element
	int nblocks = (Ndphi*Nl0)/tabDPHI_thread_x + ( ((Ndphi*Nl0)%tabDPHI_thread_x) != 0);
	dim3 dimBlock(tabDPHI_thread_x, 1);
	dim3 dimGrid(nblocks, 1);
	tabulateDPHI_gpu_kernel<<<dimGrid, dimBlock>>>(tabDPHI_d, L0diff_d, Nl0, Ndphi, convert);
	
	if(L0diff_d)hipFree(L0diff_d);
	
	*Nl0_ = Nl0;
	
	return tabDPHI_d;
}
//------------------------------------------------------------------------------------
__device__ double DPHI_gpu(double x, double y, long indexL0, double *tabDPHI, double convert, int Ndphi)
/* DOCUMENT dphi = DPHI(x,y,indexL0,rr,tabDPHI,convert) * r0^(-5./3)
 <x> & <y>         :  separation between apertures
 <indexL0>         :  index for the L0 taken into account
 <rr>              :  array of distance between apertures
 <tabDPHI>         :  array of precomputed DPHI
 <convert>         :  relation between the index on tabDPHI and (x,y)

 Computes the phase structure function for a separation (x,y).
 The r0 is not taken into account : the final result of DPHI(x,y,L0)
 has to be scaled with r0^-5/3, with r0 expressed in meters, to get
 the right value.

 SEE ALSO:
 */
{
  double r = sqrt(x * x + y * y);
  long i0 = (long) (r * convert);
  long i1 = i0 + 1;

  return ((r - (double)i0 / convert) * tabDPHI[indexL0 * Ndphi + i1]
	  + ((double)i1 / convert - r) * tabDPHI[indexL0 * Ndphi + i0]);
      
}

//------------------------------------------------------------------------------------
__device__ double cov_XX(double du, double dv, double ac, double ad, double bc, double bd, double *tabDPHI, long indexL0, double convert, int Ndphi)
 /* DOCUMENT
   Compute the XX-covariance with the distance sqrt(du2+dv2). DPHI is precomputed on tabDPHI.
 */
{
  return -DPHI_gpu(du + ac, dv, indexL0, tabDPHI, convert, Ndphi)
    + DPHI_gpu(du + ad, dv, indexL0, tabDPHI, convert, Ndphi)
    + DPHI_gpu(du + bc, dv, indexL0, tabDPHI, convert, Ndphi)
    - DPHI_gpu(du + bd, dv, indexL0, tabDPHI, convert, Ndphi);
}

//------------------------------------------------------------------------------------
__device__ double cov_YY(double du, double dv, double ac, double ad, double bc, double bd, double *tabDPHI, long indexL0, double convert, int Ndphi)
/* DOCUMENT
   Compute the YY-covariance with the distance sqrt(du2+dv2). DPHI is precomputed on tabDPHI.
 */
{ 
  return  -DPHI_gpu(du, dv + ac, indexL0, tabDPHI, convert, Ndphi)
    + DPHI_gpu(du, dv + ad, indexL0, tabDPHI, convert, Ndphi)
    + DPHI_gpu(du, dv + bc, indexL0, tabDPHI, convert, Ndphi)
    - DPHI_gpu(du, dv + bd, indexL0, tabDPHI, convert, Ndphi);
}


//------------------------------------------------------------------------------------
__device__ double cov_XY(double du, double dv, double s0, double *tabDPHI, long indexL0, double convert, int Ndphi)
/* DOCUMENT
   Compute the XY-covariance with the distance sqrt(du2+dv2). DPHI is precomputed on tabDPHI.
 */
{
  return -DPHI_gpu(du + s0, dv - s0, indexL0, tabDPHI, convert, Ndphi)
    + DPHI_gpu(du + s0, dv + s0, indexL0, tabDPHI, convert, Ndphi)
    + DPHI_gpu(du - s0, dv - s0, indexL0, tabDPHI, convert, Ndphi)
    - DPHI_gpu(du - s0, dv + s0, indexL0, tabDPHI, convert, Ndphi);
}

//------------------------------------------------------------------------------------
__global__ void subposition_gpu_kernel(long Nw, long Nsubap, long Nlayer, double *alphaX, double *alphaY,
				       double *h, double *GsAlt, long *Nssp, double *diamPup, double *thetaML,
				       long *ioff, double *X, double *Y, double *XPup, double *YPup, 
				       double *u, double *v)
{
  const int tx = threadIdx.x;
  const int ty = blockIdx.x;
	
  const int tid = ty * blockDim.x + tx;
  long i;
  long n;
  long l;
  const double rad = 3.14159265358979323846 / 180.;

  if(tid >= (Nw * Nsubap * Nlayer) ) return;
	
  l = tid / (Nw * Nsubap);

  const int pos = tid - l * (Nsubap * Nw);

  i = pos / Nw;
  n = pos - i * Nw;

  //tid = n + i * Nw + l * Nw * Nsubap

  const double dX = alphaX[n] * h[l];
  const double dY = alphaY[n] * h[l];
    
  const double rr = 1. - h[l] * GsAlt[n];
    
  const long nssp = Nssp[n];
    
  //magnification factor
  const double G = diamPup[n] / (double) (nssp);
    
  //rotation angle
  const double th = thetaML[n] * rad;

  //taking magnification factor into account
  const double xtp = X[ioff[n] + i] * G;
  const double ytp = Y[ioff[n] + i] * G;
    
  //taking rotation into account
  double uu = xtp * cos(th) - ytp * sin(th);
  double vv = xtp * sin(th) + ytp * cos(th);
    
  //taking pupil offset into account
  uu += XPup[n];
  vv += YPup[n];
    
  //Projection onto  the layer
  u[tid] = uu * rr + dX;
  v[tid] = vv * rr + dY;
}

//------------------------------------------------------------------------------------
//extern "C"
void subap_position_gpu(struct tomo_struct tomo, double *u_d, double *v_d)
//void subap_position_gpu(struct tomo_struct tomo, double ***u, double ***v)
/* DOCUMENT DOCUMENT         subap_position(tomo, u, v)
   <tomo>                : structure with all the needed information.
   <u> and <v>           : 3d arrays containing the sub-apertures projected coordinates onto all the layers. u[0][2][1] is the X-coordinate of the subap 2 of the WFS 0 on the layer 1.

   Computes the projected coordinates of all subapertures  projected onto all the layer
 */
{
  hipError_t e;
  long ioff[tomo.Nw];
  ioff[0] = 0;
  for (int i=1;i<tomo.Nw;i++) ioff[i] = ioff[i-1] + tomo.Nsubap[i-1];

  long* ioff_d;
  e = hipMalloc((void**)&ioff_d, tomo.Nw*sizeof(long));
  process_error(e, "alloc gpu ioff_d");
  e = hipMemcpy(ioff_d, ioff, tomo.Nw*sizeof(double), hipMemcpyHostToDevice);
  process_error(e, "copy gpu ioff_d");

  double *alphaX_d;
  e = hipMalloc((void**)&alphaX_d, tomo.Nw*sizeof(double));
  process_error(e, "alloc gpu alphaX_d");
  e = hipMemcpy(alphaX_d, tomo.alphaX, tomo.Nw*sizeof(double), hipMemcpyHostToDevice);
  process_error(e, "copy gpu alphaX_d");

  double *alphaY_d;
  e = hipMalloc((void**)&alphaY_d, tomo.Nw*sizeof(double));
  process_error(e, "alloc gpu alphaY_d");
  e = hipMemcpy(alphaY_d, tomo.alphaY, tomo.Nw*sizeof(double), hipMemcpyHostToDevice);
  process_error(e, "copy gpu alphaY_d");

  double *h_d;
  e = hipMalloc((void**)&h_d, tomo.Nlayer*sizeof(double));
  process_error(e, "alloc gpu h_d");
  e = hipMemcpy(h_d, tomo.h, tomo.Nlayer*sizeof(double), hipMemcpyHostToDevice);
  process_error(e, "copy gpu h_d");

  double *GsAlt_d;
  e = hipMalloc((void**)&GsAlt_d, tomo.Nw*sizeof(double));
  process_error(e, "alloc gpu GsAlt_d");
  e = hipMemcpy(GsAlt_d, tomo.GsAlt, tomo.Nw*sizeof(double), hipMemcpyHostToDevice);
  process_error(e, "copy gpu GsAlt_d");

  long *Nssp_d;
  e = hipMalloc((void**)&Nssp_d, tomo.Nw*sizeof(long));
  process_error(e, "alloc gpu Nssp_d");
  e = hipMemcpy(Nssp_d, tomo.Nssp, tomo.Nw*sizeof(long), hipMemcpyHostToDevice);
  process_error(e, "copy gpu Nssp_d");

  double *diamPup_d;
  e = hipMalloc((void**)&diamPup_d, tomo.Nw*sizeof(double));
  process_error(e, "alloc gpu diamPup_d");
  e = hipMemcpy(diamPup_d, tomo.diamPup, tomo.Nw*sizeof(double), hipMemcpyHostToDevice);
  process_error(e, "copy gpu diamPup_d");

  double *thetaML_d;
  e = hipMalloc((void**)&thetaML_d, tomo.Nw*sizeof(double));
  process_error(e, "alloc gpu thetaML_d");
  e = hipMemcpy(thetaML_d, tomo.thetaML, tomo.Nw*sizeof(double), hipMemcpyHostToDevice);
  process_error(e, "copy gpu thetaML_d");

  double *X_d;
  e = hipMalloc((void**)&X_d, tomo.Nx*sizeof(double));
  process_error(e, "alloc gpu X_d");
  e = hipMemcpy(X_d, tomo.X, tomo.Nx*sizeof(double), hipMemcpyHostToDevice);
  process_error(e, "copy gpu X_d");

  double *Y_d;
  e = hipMalloc((void**)&Y_d, tomo.Nx*sizeof(double));
  process_error(e, "alloc gpu Y_d");
  e = hipMemcpy(Y_d, tomo.Y, tomo.Nx*sizeof(double), hipMemcpyHostToDevice);
  process_error(e, "copy gpu Y_d");

  double *XPup_d;
  e = hipMalloc((void**)&XPup_d, tomo.Nw*sizeof(double));
  process_error(e, "alloc gpu XPup_d");
  e = hipMemcpy(XPup_d, tomo.XPup, tomo.Nw*sizeof(double), hipMemcpyHostToDevice);
  process_error(e, "copy gpu XPup_d");

  double *YPup_d;
  e = hipMalloc((void**)&YPup_d, tomo.Nw*sizeof(double));
  process_error(e, "alloc gpu YPup_d");
  e = hipMemcpy(YPup_d, tomo.YPup, tomo.Nw*sizeof(double), hipMemcpyHostToDevice);
  process_error(e, "copy gpu YPup_d");

  int msize = tomo.Nlayer * tomo.Nw * tomo.Nsubap[0];
  int nblocks = msize / tabDPHI_thread_x + ( ( msize % tabDPHI_thread_x) != 0);
  dim3 dimBlock(tabDPHI_thread_x, 1);
  dim3 dimGrid(nblocks, 1);
  subposition_gpu_kernel<<<dimGrid, dimBlock>>>(tomo.Nw, tomo.Nsubap[0], tomo.Nlayer, alphaX_d, alphaY_d,
						h_d, GsAlt_d, Nssp_d, diamPup_d, thetaML_d, ioff_d, X_d, 
						Y_d, XPup_d, YPup_d, u_d, v_d);
  
  if (ioff_d) hipFree(ioff_d);
  if (alphaX_d) hipFree(alphaX_d);
  if (alphaY_d) hipFree(alphaY_d);
  if (h_d) hipFree(h_d);
  if (GsAlt_d) hipFree(GsAlt_d);
  if (Nssp_d) hipFree(Nssp_d);
  if (diamPup_d) hipFree(diamPup_d);
  if (thetaML_d) hipFree(thetaML_d);
  if (X_d) hipFree(X_d);
  if (Y_d) hipFree(Y_d);
  if (XPup_d) hipFree(XPup_d);
  if (YPup_d) hipFree(YPup_d);
}



//============================================================================================
//============================= GENERATION KERNEL ============================================
//============================================================================================
__device__ double compute_element(int ipos, int jpos, int *tab_wfs, int* tab_subap, int* tab_xy, double convert,
				  double *sspSizeL, long *Nssp, double *u, double *v, double pasDPHI,double *tabDPHI, 
				  long *indexL0, double *cn2, int Ndphi, int Nw, int Nlayer, int Nsubap,
				  int type_mat, double teldiam)
{
	/* *** Covariance matrix per-element generation ***
	*   Arguments
	*   =========
	*	ipos:		Integer: global x-coordinate of the element w.r.t. the entire matrix
	*	jpos:		Integer: global y-coordinate of the element w.r.t. the entire matrix
	*/
	
	// for now return a dummy value
  
  const double lambda2 = 0.00026942094446267851;
  //WFS m
  int m = tab_wfs[ipos];
  if (type_mat == 3) m = Nw-1;
  //WFS n
  int n = tab_wfs[jpos];
  if (type_mat == 2) n = Nw-1;
  //subap i
  int i = tab_subap[ipos];
  //subap j
  int j = tab_subap[jpos];
  //xy i
  int xy_i = tab_xy[ipos];
  //xy j
  int xy_j = tab_xy[jpos];
  
  const double sspSizem = teldiam / Nssp[m];
  const double sspSizen = teldiam / Nssp[n];
  
  const double kk = lambda2 / (sspSizem * sspSizen);
    
  int type = xy_i * 2 + xy_j;

  //Layer l
  double covar = 0.0;
  #pragma unroll
  for (int l = 0; l < Nlayer; l++) 
  {
    const double sspSizeml = sspSizeL[m * Nlayer + l];
    const double sspSizenl = sspSizeL[n * Nlayer + l];
    //test if the altitude layers is not higher than the LGS altitude
    if ((sspSizeml > 0) && (sspSizenl > 0)) 
    {
      const int pos1 = m + i * Nw + l * Nw * Nsubap;
      const int pos2 = n + j * Nw + l * Nw * Nsubap;
      const double du = u[pos1] - u[pos2];	      
      const double dv =  v[pos1] - v[pos2];
      
      const double s1 = sspSizeml * 0.5;
      const double s2 = sspSizenl * 0.5;
      
      const double ac = s1 - s2;
      const double ad = s1 + s2;
      const double bc = -ad;   // initially -s1-s2;
      const double bd = -ac;   // initially -s1+s2;

      if (type == 0) covar += 0.5 * pasDPHI * cov_XX(du,dv,ac,ad,bc,bd,tabDPHI,indexL0[l],convert,Ndphi) * kk * cn2[l];
      else if (type == 3) covar += 0.5 * pasDPHI * cov_YY(du,dv,ac,ad,bc,bd,tabDPHI,indexL0[l],convert,Ndphi) * kk * cn2[l];
      else //if ((type == 1) || (type == 2)) 
      {
      	const double s0 = sqrt(s1 * s1 + s2 * s2); //half size of the subaperture equivalent to a convolution by s1 and s2
      	const double dd = (s1 > s2) ? 1. - s2 / s1 : 1. - s1 / s2; // Nono's style ....
      	covar += 0.25 * pasDPHI * cov_XY(du,dv,s0,tabDPHI,indexL0[l],convert,Ndphi) * kk * cn2[l] * (1. - dd * dd);
      }
    }
  }
  return (double)covar; 
}

__device__ double compute_element_ts(int ipos, int jpos, double convert, double *X, double *Y, 
				     long *Nssp, double pasDPHI, double *tabDPHI, long *indexL0, double *cn2, 
				     int Ndphi, int Nw, int Nlayer, int Nsubap, double teldiam)
{
	/* *** Covariance matrix per-element generation ***
	*   Arguments
	*   =========
	*	ipos:		Integer: global x-coordinate of the element w.r.t. the entire matrix
	*	jpos:		Integer: global y-coordinate of the element w.r.t. the entire matrix
	*/
	
	// for now return a dummy value
  
  const double lambda2 = 0.00026942094446267851;
  //WFS Nw-1
   //subap i
  int i = ipos < Nsubap ? ipos : ipos - Nsubap;
  //subap j
  int j = jpos < Nsubap ? jpos : jpos - Nsubap;
  //xy i
  int xy_i = ipos < Nsubap ? 0 : 1;
  //xy j
  int xy_j = jpos < Nsubap ? 0 : 1;
  
  const double sspSize = teldiam / Nssp[Nw-1];
  
  const double kk = lambda2 / (sspSize * sspSize);
    
  int type = xy_i * 2 + xy_j;

  const double s = sspSize * 0.5;
      
  const double ac = 0.0;
  const double ad = 2.0 * s;
  const double bc = -ad;   
  const double bd = 0.0;   
  const double du = X[Nsubap*(Nw-1)+i] - X[Nsubap*(Nw-1)+j];	      
  const double dv = Y[Nsubap*(Nw-1)+i] - Y[Nsubap*(Nw-1)+j];
  //const double du = X[Nw-1 + i * Nw] - X[Nw-1 + j * Nw];	      
  //const double dv = Y[Nw-1 + i * Nw] - Y[Nw-1 + j * Nw];

//Layer l
  double covar = 0.0;
  #pragma unroll
  for (int l = 0; l < Nlayer; l++) 
  {
     //test if the altitude layers is not higher than the LGS altitude
    if (sspSize > 0) 
    {
      if (type == 0) covar += 0.5 * pasDPHI * cov_XX(du,dv,ac,ad,bc,bd,tabDPHI,indexL0[l],convert,Ndphi) * 
		       kk * cn2[l];
      else if (type == 3) covar += 0.5 * pasDPHI * cov_YY(du,dv,ac,ad,bc,bd,tabDPHI,indexL0[l],convert,Ndphi) * 
			    kk * cn2[l];
      else 
      {
      	const double s0 = 1.41421*s; //half size of the subaperture equivalent to a convolution by s1 and s2
      	const double dd = 0;
      	covar += 0.25 * pasDPHI * cov_XY(du,dv,s0,tabDPHI,indexL0[l],convert,Ndphi) * 
	  kk * cn2[l] * (1. - dd * dd);
      }
    }
  }
  return (double)covar; 
}


__global__ void matcov_gpu_kernel(double* data, int nrows, int ncols, int xoffset, int yoffset, int lda,
				  int *tab_wfs, int* tab_subap, int* tab_xy, double convert, double *sspSizeL, 
				  long *Nssp, double *u, double *v, double pasDPHI,double *tabDPHI, long *indexL0, 
				  double *cn2, int Ndphi, int Nw, int Nlayer, int Nsubap, int type_mat, double teldiam)
{
  /* *** covariance matrix generation kernel ***
   *	The kernel generates the element values in a given matrix/submatrix
   *   The generation function can be any function, as long as each element 
   *   can be computed both individually and independently
   *
   *	see argument description in the kernel driver
   */
	
  // local thread coordinates w.r.t. thread block
  const int tx_ = threadIdx.x;
  const int ty_ = threadIdx.y;
	
  // local thread block coordinates w.r.t. kernel grid
  const int bx_ = blockIdx.x;
  const int by_ = blockIdx.y;
	
  // local coordinates of the element w.r.t. submatrix
  int lx = bx_ * blockDim.x + tx_;
  int ly = by_ * blockDim.y + ty_;
	
  // global coordinates of the elemnt w.r.t. the entire matrix
  int gx = lx + xoffset;
  int gy = ly + yoffset;
	
  // out-of-bound threads should terminate
  if( (lx >= nrows) || (ly >= ncols) ) return;
	
  // Advance the data pointer accordingly
  data += ly * lda + lx;
	
  if ((type_mat == 3) || (gx <= gy)) {
    // call the generation function
    data[0] = compute_element(gx, gy, tab_wfs, tab_subap, tab_xy,convert,sspSizeL,Nssp,u,v,pasDPHI,tabDPHI,
			      indexL0,cn2,Ndphi,Nw,Nlayer,Nsubap,type_mat,teldiam);
    //printf("gx = %d, gy = %d ----- %.2f \n", gx, gy, data[0]);
  } 
}

__global__ void matts_gpu_kernel(double* data, int nrows, int ncols, int xoffset, int yoffset, int lda,
				  double convert, double *X, double *Y, long *Nssp, double pasDPHI,double *tabDPHI, 
				 long *indexL0, double *cn2, int Ndphi, int Nw, int Nlayer, int Nsubap, double teldiam)
{
  /* *** covariance matrix generation kernel ***
   *	The kernel generates the element values in a given matrix/submatrix
   *   The generation function can be any function, as long as each element 
   *   can be computed both individually and independently
   *
   *	see argument description in the kernel driver
   */
	
  // local thread coordinates w.r.t. thread block
  const int tx_ = threadIdx.x;
  const int ty_ = threadIdx.y;
	
  // local thread block coordinates w.r.t. kernel grid
  const int bx_ = blockIdx.x;
  const int by_ = blockIdx.y;
	
  // local coordinates of the element w.r.t. submatrix
  int lx = bx_ * blockDim.x + tx_;
  int ly = by_ * blockDim.y + ty_;
	
  // global coordinates of the elemnt w.r.t. the entire matrix
  int gx = lx + xoffset;
  int gy = ly + yoffset;
	
  // out-of-bound threads should terminate
  if( (lx >= nrows) || (ly >= ncols) ) return;
	
  // Advance the data pointer accordingly
  data += ly * lda + lx;
	
    // call the generation function
    data[0] = compute_element_ts(gx, gy, convert,X, Y,Nssp,pasDPHI,tabDPHI,
			      indexL0,cn2,Ndphi,Nw,Nlayer,Nsubap,teldiam);
    //printf("gx = %d, gy = %d ----- %.2f \n", gx, gy, data[0]);
}

__global__ void matcov_gpu_kernel_copy(double* data, int nrows, int ncols, int xoffset, int yoffset, int lda)
{
  /* *** covariance matrix generation kernel ***
   *	The kernel generates the element values in a given matrix/submatrix
   *   The generation function can be any function, as long as each element 
   *   can be computed both individually and independently
   *
   *	see argument description in the kernel driver
   */
	
  // local thread coordinates w.r.t. thread block
  const int tx_ = threadIdx.x;
  const int ty_ = threadIdx.y;
	
  // local thread block coordinates w.r.t. kernel grid
  const int bx_ = blockIdx.x;
  const int by_ = blockIdx.y;
	
  // local coordinates of the element w.r.t. submatrix
  int lx = bx_ * blockDim.x + tx_;
  int ly = by_ * blockDim.y + ty_;
	
  // global coordinates of the elemnt w.r.t. the entire matrix
  int gx = lx + xoffset;
  int gy = ly + yoffset;
	
  // out-of-bound threads should terminate
  if( (lx >= nrows) || (ly >= ncols) ) return;
	
  // Advance the data pointer accordingly
  //data += ly * lda + lx;
	
  if (gx > gy) {
    // call the generation function
    data[ly * lda + lx] = data[ly + lx * lda];
    //printf("gx = %d, gy = %d ----- %.2f \n", gx, gy, data[0]);
  }
}

//extern "C"
void matcov_gpu(double* data, int nrows, int ncols, int xoffset, int yoffset, int lda, struct tomo_struct tomo)
{
	/* *** matcov gpu kernel driver ***
	*  Arguments
	*  ==========
	*  data		double pointer: A pointer to the matrix/submatrix to be generated. It  
	*  			should always point to the first element in a matrix/submatrix
	*
	*  nrows	integer: The number of rows of the matrix/submatrix to be generated	
	*
	*  ncols	integer: The number of columns of the matrix/submatrix to be generated
	*
	*  xoffset	integer: The x-offset of the submatrix, must be zero if the entire matrix
	*			is generated. Its the x-coordinate of the first element in the matrix/submatrix
	*
	*  yoffset  integer: The y-offset of the submatrix, must be zero if the entire matrix
	*			is generated. Its the y-coordinate of the first element in the matrix/submatrix
	*
	*  lda		integer: The leading dimension of the matrix/submatrix
	*/
	
  hipError_t e;
  // %%%%%%% Pre-computation of DPHI %%%%%%%%%%
  //Computes an array of DPHI (tabDPHI) for an array of subaperture distance rr for each DIFFERENT L0
  const long Nw = tomo.Nw;
  const long Nlayer = tomo.Nlayer;
  const double crmax = tomo.rmax;
  const double pasDPHI = 1./tomo.pasDPHI; //inverse du pas de rr
  const long Ndphi = floor(crmax*pasDPHI)+1;
  const double convert = (double)(Ndphi-1)/(crmax+1./pasDPHI);

  long indexL0[Nlayer]; //link between index in L0 and index in L0diff

  double *tabDPHI_d;
  int Nl0_; // used to know the size of the array
  tabDPHI_d = tabulateDPHI_gpu(tomo, Ndphi, indexL0, (int*)&Nl0_,convert);

  long *indexL0_d;
  //printf("sizeof indexL0 is %.2f KB\n", Nlayer*sizeof(long)/1024.0);
  e = hipMalloc((void**)&indexL0_d, Nlayer*sizeof(long));
  process_error(e, "alloc gpu indexL0_d");
  e = hipMemcpy(indexL0_d, indexL0, Nlayer*sizeof(long), hipMemcpyHostToDevice);
  process_error(e, "copy gpu indexL0_d");

  // %%%%%%% Computation of the sub-apertures positions and sizes %%%%%%%%%%%
 // u, v :arrays containing all the sub-apertures coordinates of all WFS, one after the other
  // u[0][1][3] is the X-coordinate of subap number 3 of wfs number 0 at altitude 3
  double* u_d;
  //printf("sizeof u is %.2f KB\n", Nlayer*tomo.Nsubap[0]*Nw*sizeof(double)/1024.0);
  e = hipMalloc((void**)&u_d, Nlayer*tomo.Nsubap[0]*Nw*sizeof(double));
  process_error(e, "alloc gpu u_d");
  double* v_d;
  e = hipMalloc((void**)&v_d, Nlayer*tomo.Nsubap[0]*Nw*sizeof(double));
  process_error(e, "alloc gpu v_d");

  //Computes  u and v
  subap_position_gpu(tomo, u_d, v_d);
 

  double *sspSizeL = (double *)malloc(sizeof(double)*Nw*Nlayer);
  for (int cc = 0; cc < Nw * Nlayer; cc++) {
    int n = cc / Nlayer;
    int l = cc - n * Nlayer;
    sspSizeL[cc] = tomo.sspSize[n] * (1. - tomo.GsAlt[n] * tomo.h[l]);
  }

  double *sspSizeL_d;
  //printf("sizeof sspSizeL is %.2f KB\n", Nw*Nlayer*sizeof(double)/1024.0);
  e = hipMalloc((void**)&sspSizeL_d, Nw*Nlayer*sizeof(double));
  process_error(e, "alloc gpu sspSizeL_d");
  e = hipMemcpy(sspSizeL_d, sspSizeL, Nw*Nlayer*sizeof(double), hipMemcpyHostToDevice);
  process_error(e, "copy gpu sspSizeL_d");

  int *tab_wfs;
  tab_wfs = (int*)malloc(nrows*sizeof(int));
  int *tab_subap;
  tab_subap = (int*)malloc(nrows*sizeof(int));
  int *tab_xy;
  tab_xy = (int*)malloc(nrows*sizeof(int));
 
  long ts = Nw - 1;//Truth sensor : ts
  int cpt = 0;
  for (int cc=0;cc<Nw;cc++) {
    if (cc != ts) {
      int nslps = tomo.Nsubap[cc]*2;
      for (int ccc=0;ccc<nslps;ccc++) {
	if (cc > ts) tab_wfs[ccc+cpt] = cc - 1;
	else tab_wfs[ccc+cpt] = cc;
	if (ccc < nslps/2) {
	  tab_subap[ccc+cpt] = ccc;
	  tab_xy[ccc+cpt] = 0;
	} else {
	  tab_subap[ccc+cpt] = ccc - nslps/2;
	  tab_xy[ccc+cpt] = 1;
	}
      }
      cpt += nslps;
    }
  }

	//for(int ah = 0; ah  < nrows; ah++)
	//	printf("[%5d]: tab_wfs = %6d, tab_subap = %6d, tab_xy = %6d\n", ah, tab_wfs[ah], tab_subap[ah], tab_xy[ah]);
  int *tab_wfs_d;
  //printf("sizeof tab_wfs is %.2f KB\n", nrows*sizeof(int)/1024.0);
  e = hipMalloc((void**)&tab_wfs_d, nrows*sizeof(int));
  process_error(e, "alloc gpu tab_wfs_d");
  e = hipMemcpy(tab_wfs_d, tab_wfs, nrows*sizeof(int), hipMemcpyHostToDevice);
  process_error(e, "copy gpu tab_wfs_d");

  int *tab_subap_d;
  //printf("sizeof tab_subap is %.2f KB\n", nrows*sizeof(int)/1024.0);
  e = hipMalloc((void**)&tab_subap_d, nrows*sizeof(int));
  process_error(e, "alloc gpu tab_subap_d");
  e = hipMemcpy(tab_subap_d, tab_subap, nrows*sizeof(int), hipMemcpyHostToDevice);
  process_error(e, "copy gpu tab_subap_d");

  int *tab_xy_d;
  //printf("sizeof tab_xy is %.2f KB\n", nrows*sizeof(int)/1024.0);
  e = hipMalloc((void**)&tab_xy_d, nrows*sizeof(int));
  process_error(e, "alloc gpu tab_xy_d");
  e = hipMemcpy(tab_xy_d, tab_xy, nrows*sizeof(int), hipMemcpyHostToDevice);
  process_error(e, "copy gpu tab_xy_d");

  double *cn2_d;
  //printf("sizeof cn2_d is %.2f KB\n", Nlayer*sizeof(double)/1024.0);
  e = hipMalloc((void**)&cn2_d, Nlayer*sizeof(double));
  process_error(e, "alloc gpu cn2_d");
  e = hipMemcpy(cn2_d, tomo.cn2, Nlayer*sizeof(double), hipMemcpyHostToDevice);
  process_error(e, "copy gpu cn2_d");

  long *Nssp_d;
  e = hipMalloc((void**)&Nssp_d, Nw*sizeof(long));
  process_error(e, "alloc gpu Nssp_d");
  e = hipMemcpy(Nssp_d, tomo.Nssp, Nw*sizeof(long), hipMemcpyHostToDevice);
  process_error(e, "copy gpu Nssp_d");

  int nbx = nrows / matcov_thread_x + (nrows%matcov_thread_x != 0);
  int nby = ncols / matcov_thread_y + (ncols%matcov_thread_y != 0);

  dim3 dimBlock(matcov_thread_x, matcov_thread_y);
  dim3 dimGrid(nbx, nby);
  const long Nsubap = tomo.Nsubap[0];
  int type_mat = tomo.part;
  
  //printf("Nlayer = %d \n", Nlayer);
  
  matcov_gpu_kernel<<<dimGrid, dimBlock>>>(data, nrows, ncols, xoffset, yoffset, lda, tab_wfs_d, tab_subap_d, tab_xy_d,
					   convert,sspSizeL_d,Nssp_d,u_d,v_d,pasDPHI,tabDPHI_d,indexL0_d,cn2_d,
					   Ndphi,Nw,Nlayer,Nsubap,type_mat,tomo.DiamTel);
  
  matcov_gpu_kernel_copy<<<dimGrid, dimBlock>>>(data, nrows, ncols, xoffset, yoffset, lda);
  


  if (sspSizeL) free(sspSizeL);
  if (tab_wfs) free(tab_wfs);
  if (tab_subap) free(tab_subap);
  if (tab_xy) free(tab_xy);

  if (sspSizeL_d) hipFree(sspSizeL_d);
  if (tab_wfs_d) hipFree(tab_wfs_d);
  if (tab_subap_d) hipFree(tab_subap_d);
  if (tab_xy_d) hipFree(tab_xy_d);
  if (indexL0_d) hipFree(indexL0_d);
  if (tabDPHI_d) hipFree(tabDPHI_d);
  if (cn2_d) hipFree(cn2_d);
  if (Nssp_d) hipFree(Nssp_d);
  if (u_d) hipFree(u_d);;
  if (v_d) hipFree(v_d);;
}

//======================================================================================================
//                        V3
//======================================================================================================


void init_tomo_gpu(struct tomo_gpu_struct *tomo_gpu, struct tomo_struct tomo){
  hipError_t e;

  e = hipMalloc((void**)&(tomo_gpu->indexL0_d), tomo.Nlayer*sizeof(long));
  process_error(e, "alloc gpu indexL0_d");

  e = hipMalloc((void**)&(tomo_gpu->u_d), tomo.Nlayer*tomo.Nsubap[0]*tomo.Nw*sizeof(double));
  process_error(e, "alloc gpu u_d");

  e = hipMalloc((void**)&(tomo_gpu->v_d), tomo.Nlayer*tomo.Nsubap[0]*tomo.Nw*sizeof(double));
  process_error(e, "alloc gpu v_d");

  e = hipMalloc((void**)&(tomo_gpu->sspSizeL_d), tomo.Nw*tomo.Nlayer*sizeof(double));
  process_error(e, "alloc gpu sspSizeL_d");

  e = hipMalloc((void**)&(tomo_gpu->cn2_d), tomo.Nw*tomo.Nlayer*sizeof(double));
  process_error(e, "alloc gpu cn2_d");

  e = hipMalloc((void**)&(tomo_gpu->h_d), tomo.Nlayer*sizeof(double));
  process_error(e, "alloc gpu h_d");


  e = hipMalloc((void**)&(tomo_gpu->Nssp_d), tomo.Nw*sizeof(long));
  process_error(e, "alloc gpu Nssp_d");

  e = hipMalloc((void**)&(tomo_gpu->ioff_d), tomo.Nw*sizeof(long));
  process_error(e, "alloc gpu ioff_d");

  e = hipMalloc((void**)&(tomo_gpu->alphaX_d), tomo.Nw*sizeof(double));
  process_error(e, "alloc gpu alphaX_d");

  e = hipMalloc((void**)&(tomo_gpu->alphaY_d), tomo.Nw*sizeof(double));
  process_error(e, "alloc gpu alphaY_d");

  e = hipMalloc((void**)&(tomo_gpu->GsAlt_d), tomo.Nw*sizeof(double));
  process_error(e, "alloc gpu GsAlt_d");

  e = hipMalloc((void**)&(tomo_gpu->diamPup_d), tomo.Nw*sizeof(double));
  process_error(e, "alloc gpu diamPup_d");

  e = hipMalloc((void**)&(tomo_gpu->thetaML_d), tomo.Nw*sizeof(double));
  process_error(e, "alloc gpu thetaML_d");

  e = hipMalloc((void**)&(tomo_gpu->X_d), tomo.Nx*sizeof(double));
  process_error(e, "alloc gpu X_d");

  e = hipMalloc((void**)&(tomo_gpu->Y_d), tomo.Nx*sizeof(double));
  process_error(e, "alloc gpu Y_d");

  e = hipMalloc((void**)&(tomo_gpu->XPup_d), tomo.Nw*sizeof(double));
  process_error(e, "alloc gpu XPup_d");

  e = hipMalloc((void**)&(tomo_gpu->YPup_d), tomo.Nw*sizeof(double));
  process_error(e, "alloc gpu YPup_d");

  //printf("dims : %d %d %d\n",tomo.Nsubap[tomo.Nw-1],tomo.Nsubap[0],tomo.Nx);
  /*
  e = hipMalloc((void**)&(tomo_gpu->Cmm_d), tomo.Nw*tomo.Nsubap[0]*2*tomo.Nw*tomo.Nsubap[0]*2*sizeof(double));
  process_error(e, "alloc gpu YPup_d");

  e = hipMalloc((void**)&(tomo_gpu->Cpm_d), tomo.Nsubap[0]*2*tomo.Nw*tomo.Nsubap[0]*2*sizeof(double));
  process_error(e, "alloc gpu YPup_d");

  e = hipMalloc((void**)&(tomo_gpu->R_d), tomo.Nsubap[0]*2*tomo.Nw*tomo.Nsubap[0]*2*sizeof(double));
  process_error(e, "alloc gpu YPup_d");
  */
  
  tomo_gpu->L0diff_d = NULL;
  tomo_gpu->tabDPHI_d = NULL;
  
  e = hipStreamCreate(&(tomo_gpu->matcov_stream));
  process_error(e, "create matcov stream");

}

void free_tomo_gpu(struct tomo_gpu_struct *tomo_gpu){
  hipError_t e;

  if (tomo_gpu->u_d) e = hipFree(tomo_gpu->u_d);
  process_error(e, "free gpu u_d");

  if (tomo_gpu->v_d) e = hipFree(tomo_gpu->v_d);
  process_error(e, "free gpu v_d");

  if (tomo_gpu->sspSizeL_d) e = hipFree(tomo_gpu->sspSizeL_d) ;
  process_error(e, "free gpu sspSizeL_d");

  if (tomo_gpu->cn2_d) e = hipFree(tomo_gpu->cn2_d);
  process_error(e, "free gpu cn2_d");

  if (tomo_gpu->h_d) e = hipFree(tomo_gpu->h_d);
  process_error(e, "free gpu h_d");

  if (tomo_gpu->indexL0_d) e = hipFree(tomo_gpu->indexL0_d);
  process_error(e, "free gpu indexL0_d");


  if (tomo_gpu->Nssp_d) e = hipFree(tomo_gpu->Nssp_d);
  process_error(e, "free gpu Nssp_d");

  if (tomo_gpu->ioff_d) e = hipFree(tomo_gpu->ioff_d);
  process_error(e, "free gpu ioff_d");

  if (tomo_gpu->alphaX_d) e = hipFree(tomo_gpu->alphaX_d);
  process_error(e, "free gpu alphaX_d");

  if (tomo_gpu->alphaY_d) e = hipFree(tomo_gpu->alphaY_d);
  process_error(e, "free gpu alphaY_d");

  if (tomo_gpu->GsAlt_d) e = hipFree(tomo_gpu->GsAlt_d);
  process_error(e, "free gpu GsAlt_d");

  if (tomo_gpu->diamPup_d) e = hipFree(tomo_gpu->diamPup_d);
  process_error(e, "free gpu diamPup_d");

  if (tomo_gpu->thetaML_d) e = hipFree(tomo_gpu->thetaML_d);
  process_error(e, "free gpu thetaML_d");

  if (tomo_gpu->X_d) e = hipFree(tomo_gpu->X_d);
  process_error(e, "free gpu X_d");

  if (tomo_gpu->Y_d) e = hipFree(tomo_gpu->Y_d);
  process_error(e, "free gpu Y_d");

  if (tomo_gpu->XPup_d) e = hipFree(tomo_gpu->XPup_d);
  process_error(e, "free gpu XPup_d");

  if (tomo_gpu->YPup_d) e = hipFree(tomo_gpu->YPup_d);
  process_error(e, "free gpu YPup_d");

  /*
  if (tomo_gpu->Cmm_d) e = hipFree(tomo_gpu->Cmm_d);
  process_error(e, "free gpu YPup_d");

  if (tomo_gpu->Cpm_d) e = hipFree(tomo_gpu->Cpm_d);
  process_error(e, "free gpu YPup_d");

  if (tomo_gpu->R_d) e = hipFree(tomo_gpu->R_d);
  process_error(e, "free gpu YPup_d");
  */

  if ((tomo_gpu->tabDPHI_d) != NULL) e = hipFree(tomo_gpu->tabDPHI_d);
  process_error(e, "free gpu tabDPHI_d");

  if ((tomo_gpu->L0diff_d) != NULL) e = hipFree(tomo_gpu->L0diff_d);
  process_error(e, "free gpu L0diff_d");
  
  // destroy matcov stream
  e = hipStreamDestroy(tomo_gpu->matcov_stream);
  process_error(e, "destroy matcov stream");
}

//------------------------------------------------------------------------------------
void tab_dphi_gpu(double *tab_dphi, struct tomo_struct tomo, struct tomo_gpu_struct *tomo_gpu, long Ndphi, double *L0diff_d, int Nl0, double convert)
//void tabulateDPHI_gpu(double* tabDPHI_d, double* rr_d,struct tomo_struct tomo, long Ndphi, long *indexL0_h)
/* DOCUMENT tabDPHI = tabulateDPHI(rr,tomo,Ndphi, indexL0)
 <tomo>            :  structure with all the needed information
 <Ndphi>           :  size of rr
 <indexL0>         :  link between the index of the studied layer and the index of the precomputed one. 

 Computes the phase structure function for a separation rr(x,y).
 The r0 is not taken into account : the final result of DPHI(x,y,L0)
 has to be scaled with r0^-5/3, with r0 expressed in meters, to get
 the right value.

 Computes the phase structure for each different L0 and give a array (indexL0) to link the index of the layer i and the index of tabDPHI : for the layer l, DPHI = DPHI( du, dv, indexL0[l],rr,tabDPHI, convert).
 SEE ALSO: DPHI
 */
{
  // Assume one thread per element
  int nblocks = (Ndphi*Nl0)/tabDPHI_thread_x + ( ((Ndphi*Nl0)%tabDPHI_thread_x) != 0);
  dim3 dimBlock(tabDPHI_thread_x, 1);
  dim3 dimGrid(nblocks, 1);

  tabulateDPHI_gpu_kernel<<<dimGrid, dimBlock, 0, tomo_gpu->matcov_stream>>>(tab_dphi, L0diff_d, Nl0, Ndphi, convert);
  CudaCheckError();
}


//------------------------------------------------------------------------------------
//extern "C"
void sub_pos_gpu(struct tomo_gpu_struct *tomo_gpu, struct tomo_struct tomo)
//void subap_position_gpu(struct tomo_struct tomo, double ***u, double ***v)
/* DOCUMENT DOCUMENT         subap_position(tomo, u, v)
   <tomo>                : structure with all the needed information.
   <u> and <v>           : 3d arrays containing the sub-apertures projected coordinates onto all the layers. u[0][2][1] is the X-coordinate of the subap 2 of the WFS 0 on the layer 1.

   Computes the projected coordinates of all subapertures  projected onto all the layer
 */
{
  int msize = tomo.Nlayer * tomo.Nw * tomo.Nsubap[0];
  int nblocks = msize / tabDPHI_thread_x + ( ( msize % tabDPHI_thread_x) != 0);
  dim3 dimBlock(tabDPHI_thread_x, 1);
  dim3 dimGrid(nblocks, 1);
  subposition_gpu_kernel<<<dimGrid, dimBlock, 0, tomo_gpu->matcov_stream>>>(tomo.Nw, tomo.Nsubap[0], tomo.Nlayer, tomo_gpu->alphaX_d, 
						tomo_gpu->alphaY_d,tomo_gpu->h_d, tomo_gpu->GsAlt_d, 
						tomo_gpu->Nssp_d, tomo_gpu->diamPup_d, tomo_gpu->thetaML_d, 
						tomo_gpu->ioff_d, tomo_gpu->X_d, tomo_gpu->Y_d, 
						tomo_gpu->XPup_d, tomo_gpu->YPup_d, tomo_gpu->u_d, tomo_gpu->v_d);
  CudaCheckError();

}


void update_tomo_atm(struct tomo_gpu_struct *tomo_gpu,struct tomo_struct tomo) {

  hipError_t e;

  const double crmax = tomo.rmax;
  const double pasDPHI = 1./tomo.pasDPHI; //inverse du pas de rr
  const long Ndphi = floor(crmax*pasDPHI)+1;
  const double convert = (double)(Ndphi-1)/(crmax+1./pasDPHI);

  e = hipMemcpyAsync(tomo_gpu->h_d, tomo.h, tomo.Nlayer*sizeof(double), hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_error(e, "copy gpu h_d");
  e = hipMemcpyAsync(tomo_gpu->cn2_d, tomo.cn2, tomo.Nlayer*sizeof(double), hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_error(e, "copy gpu cn2_d");


  double *sspSizeL = (double *)malloc(sizeof(double)*tomo.Nw*tomo.Nlayer);
  for (int cc = 0; cc < tomo.Nw * tomo.Nlayer; cc++) {
    int n = cc / tomo.Nlayer;
    int l = cc - n * tomo.Nlayer;
    sspSizeL[cc] = tomo.sspSize[n] * (1. - tomo.GsAlt[n] * tomo.h[l]);
  }

  e = hipMemcpyAsync(tomo_gpu->sspSizeL_d, sspSizeL, tomo.Nw*tomo.Nlayer*sizeof(double), hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_error(e, "copy gpu sspSizeL_d");

  //Search the different L0 and build indexL0
  const long Nlayer = tomo.Nlayer;
  long i, j;
  int cpt = 1;
  double tmp[Nlayer];
  long indexL0[Nlayer];

  tmp[0] = tomo.L0[0];
  indexL0[0] = 0;
  
  for (i = 1; i < Nlayer; i++) {
    j = 0;
    const double l0 = tomo.L0[i];
    
    while ((j < cpt) && (tmp[j] != l0)) {j++;}
    
    indexL0[i] = j;
    
    if (j == cpt) {
      tmp[j] = l0;
      cpt++;
    }
  }
  
  e = hipMemcpyAsync((tomo_gpu->indexL0_d), indexL0, tomo.Nlayer*sizeof(long), hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_error(e, "copy gpu indexL0_d");

  tomo_gpu->Nl0 = cpt;
  double L0diff[tomo_gpu->Nl0];

  // allocate space for L0
  if ((tomo_gpu->L0diff_d) != NULL){hipFree(tomo_gpu->L0diff_d);}
  e = hipMalloc((void**)&(tomo_gpu->L0diff_d), tomo_gpu->Nl0*sizeof(double));
  process_error(e, "alloc gpu L0diff_d");
  
  for (i = 0; i < tomo_gpu->Nl0; i++)  {
    L0diff[i] = tmp[i];
  }
  
  // offload L0diff
  e = hipMemcpyAsync(tomo_gpu->L0diff_d, L0diff, tomo_gpu->Nl0*sizeof(double), hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_error(e, "offload L0diff");
  
  //précalcul de DPHI : que pour chaque différent L0
  if ((tomo_gpu->tabDPHI_d) != NULL){hipFree(tomo_gpu->tabDPHI_d);}
  e = hipMalloc((void**)&(tomo_gpu->tabDPHI_d), tomo_gpu->Nl0*Ndphi*sizeof(double));
  process_error(e, "alloc gpu tabDPHI_d");
  
  tab_dphi_gpu(tomo_gpu->tabDPHI_d, tomo, tomo_gpu, Ndphi, tomo_gpu->L0diff_d, tomo_gpu->Nl0,convert);

  // %%%%%%% Computation of the sub-apertures positions and sizes %%%%%%%%%%%
 // u, v :arrays containing all the sub-apertures coordinates of all WFS, one after the other
  // u[0][1][3] is the X-coordinate of subap number 3 of wfs number 0 at altitude 3

  //Computes  u and v
  sub_pos_gpu(tomo_gpu, tomo);
 
  if (sspSizeL) free(sspSizeL);
  hipStreamSynchronize(tomo_gpu->matcov_stream);
}

void update_tomo_sys(struct tomo_gpu_struct *tomo_gpu,struct tomo_struct tomo) {

  hipError_t e;

  long ioff[tomo.Nw];
  ioff[0] = 0;
  for (int i=1;i<tomo.Nw;i++) ioff[i] = ioff[i-1] + tomo.Nsubap[i-1];
  e = hipMemcpyAsync(tomo_gpu->ioff_d, ioff, tomo.Nw*sizeof(double), hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_error(e, "copy gpu ioff_d");

  e = hipMemcpyAsync(tomo_gpu->alphaX_d, tomo.alphaX, tomo.Nw*sizeof(double), hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_error(e, "copy gpu alphaX_d");
  e = hipMemcpyAsync(tomo_gpu->alphaY_d, tomo.alphaY, tomo.Nw*sizeof(double), hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_error(e, "copy gpu alphaY_d");

  e = hipMemcpyAsync(tomo_gpu->GsAlt_d, tomo.GsAlt, tomo.Nw*sizeof(double), hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_error(e, "copy gpu GsAlt_d");

  e = hipMemcpyAsync(tomo_gpu->Nssp_d, tomo.Nssp, tomo.Nw*sizeof(long), hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_error(e, "copy gpu Nssp_d");

  e = hipMemcpyAsync(tomo_gpu->diamPup_d, tomo.diamPup, tomo.Nw*sizeof(double), hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_error(e, "copy gpu diamPup_d");

  e = hipMemcpyAsync(tomo_gpu->XPup_d, tomo.XPup, tomo.Nw*sizeof(double), hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_error(e, "copy gpu XPup_d");
  e = hipMemcpyAsync(tomo_gpu->YPup_d, tomo.YPup, tomo.Nw*sizeof(double), hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_error(e, "copy gpu YPup_d");
  e = hipMemcpyAsync(tomo_gpu->thetaML_d, tomo.thetaML, tomo.Nw*sizeof(double), hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_error(e, "copy gpu thetaML_d");

  e = hipMemcpyAsync(tomo_gpu->X_d, tomo.X, tomo.Nx*sizeof(double), hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_error(e, "copy gpu X_d");
  e = hipMemcpyAsync(tomo_gpu->Y_d, tomo.Y, tomo.Nx*sizeof(double), hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_error(e, "copy gpu Y_d");
  
  hipStreamSynchronize(tomo_gpu->matcov_stream);
}


//extern "C"
void matcov_gpu3(double* data, int nrows, int ncols, int xoffset, int yoffset, int lda, struct tomo_struct tomo, 
		struct tomo_gpu_struct *tomo_gpu)
{
	/* *** matcov gpu kernel driver ***
	*  Arguments
	*  ==========
	*  data		double pointer: A pointer to the matrix/submatrix to be generated. It  
	*  			should always point to the first element in a matrix/submatrix
	*
	*  nrows	integer: The number of rows of the matrix/submatrix to be generated	
	*
	*  ncols	integer: The number of columns of the matrix/submatrix to be generated
	*
	*  xoffset	integer: The x-offset of the submatrix, must be zero if the entire matrix
	*			is generated. Its the x-coordinate of the first element in the matrix/submatrix
	*
	*  yoffset  integer: The y-offset of the submatrix, must be zero if the entire matrix
	*			is generated. Its the y-coordinate of the first element in the matrix/submatrix
	*
	*  lda		integer: The leading dimension of the matrix/submatrix
	*/
	
  hipError_t e;
  // %%%%%%% Pre-computation of DPHI %%%%%%%%%%
  //Computes an array of DPHI (tabDPHI) for an array of subaperture distance rr for each DIFFERENT L0
  const long Nw = tomo.Nw;
  const double crmax = tomo.rmax;
  const double pasDPHI = 1./tomo.pasDPHI; //inverse du pas de rr
  const long Ndphi = floor(crmax*pasDPHI)+1;
  const double convert = (double)(Ndphi-1)/(crmax+1./pasDPHI);

  int type_mat = tomo.part;

  int size = tomo.Nslopes - 2 * tomo.Nsubap[tomo.Nw-1];

  int *tab_wfs;
  tab_wfs = (int*)malloc(size*sizeof(int));
  int *tab_subap;
  tab_subap = (int*)malloc(size*sizeof(int));
  int *tab_xy;
  tab_xy = (int*)malloc(size*sizeof(int));
 
  long ts = Nw - 1;//Truth sensor : ts
  int cpt = 0;
  for (int cc=0;cc<Nw;cc++) {
    if (cc != ts) {
      int nslps = tomo.Nsubap[cc]*2;
      for (int ccc=0;ccc<nslps;ccc++) {
	if (cc > ts) tab_wfs[ccc+cpt] = cc - 1;
	else tab_wfs[ccc+cpt] = cc;
	if (ccc < nslps/2) {
	  tab_subap[ccc+cpt] = ccc;
	  tab_xy[ccc+cpt] = 0;
	} else {
	  tab_subap[ccc+cpt] = ccc - nslps/2;
	  tab_xy[ccc+cpt] = 1;
	}
      }
      cpt += nslps;
    }
  }

  int *tab_wfs_d;
  //printf("sizeof tab_wfs is %.2f KB\n", nrows*sizeof(int)/1024.0);
  e = hipMalloc((void**)&tab_wfs_d, size*sizeof(int));
  process_error(e, "alloc gpu tab_wfs_d");
  e = hipMemcpy(tab_wfs_d, tab_wfs, size*sizeof(int), hipMemcpyHostToDevice);
  process_error(e, "copy gpu tab_wfs_d");

  int *tab_subap_d;
  //printf("sizeof tab_subap is %.2f KB\n", nrows*sizeof(int)/1024.0);
  e = hipMalloc((void**)&tab_subap_d, size*sizeof(int));
  process_error(e, "alloc gpu tab_subap_d");
  e = hipMemcpy(tab_subap_d, tab_subap, size*sizeof(int), hipMemcpyHostToDevice);
  process_error(e, "copy gpu tab_subap_d");

  int *tab_xy_d;
  //printf("sizeof tab_xy is %.2f KB\n", nrows*sizeof(int)/1024.0);
  e = hipMalloc((void**)&tab_xy_d, size*sizeof(int));
  process_error(e, "alloc gpu tab_xy_d");
  e = hipMemcpy(tab_xy_d, tab_xy, size*sizeof(int), hipMemcpyHostToDevice);
  process_error(e, "copy gpu tab_xy_d");

  int nbx = nrows / matcov_thread_x + (nrows%matcov_thread_x != 0);
  int nby = ncols / matcov_thread_y + (ncols%matcov_thread_y != 0);

  dim3 dimBlock(matcov_thread_x, matcov_thread_y);
  dim3 dimGrid(nbx, nby);
  const long Nsubap = tomo.Nsubap[0];
  
  matcov_gpu_kernel<<<dimGrid, dimBlock, 0, tomo_gpu->matcov_stream>>>(data, nrows, ncols, xoffset, yoffset, lda, tab_wfs_d, tab_subap_d, tab_xy_d,
					   convert,tomo_gpu->sspSizeL_d,tomo_gpu->Nssp_d,tomo_gpu->u_d,tomo_gpu->v_d,
					   pasDPHI,tomo_gpu->tabDPHI_d,tomo_gpu->indexL0_d,tomo_gpu->cn2_d,
					   Ndphi,tomo.Nw,tomo.Nlayer,Nsubap,type_mat,tomo.DiamTel);
  
  if (type_mat == 1)
    matcov_gpu_kernel_copy<<<dimGrid, dimBlock>>>(data, nrows, ncols, xoffset, yoffset, lda);
  
	hipStreamSynchronize(tomo_gpu->matcov_stream);
	
	if (tab_wfs) free(tab_wfs);
	if (tab_subap) free(tab_subap);
	if (tab_xy) free(tab_xy);
	
	if (tab_wfs_d) hipFree(tab_wfs_d);
	if (tab_subap_d) hipFree(tab_subap_d);
	if (tab_xy_d) hipFree(tab_xy_d);
}

void matts_gpu(double* data, int nrows, int ncols, int xoffset, int yoffset, int lda, struct tomo_struct tomo, 
		struct tomo_gpu_struct *tomo_gpu)
{
	/* *** matcov gpu kernel driver ***
	*  Arguments
	*  ==========
	*  data		double pointer: A pointer to the matrix/submatrix to be generated. It  
	*  			should always point to the first element in a matrix/submatrix
	*
	*  nrows	integer: The number of rows of the matrix/submatrix to be generated	
	*
	*  ncols	integer: The number of columns of the matrix/submatrix to be generated
	*
	*  xoffset	integer: The x-offset of the submatrix, must be zero if the entire matrix
	*			is generated. Its the x-coordinate of the first element in the matrix/submatrix
	*
	*  yoffset  integer: The y-offset of the submatrix, must be zero if the entire matrix
	*			is generated. Its the y-coordinate of the first element in the matrix/submatrix
	*
	*  lda		integer: The leading dimension of the matrix/submatrix
	*/
	
  const long Nw = tomo.Nw;
  const double crmax = tomo.rmax;
  const double pasDPHI = 1./tomo.pasDPHI; //inverse du pas de rr
  const long Ndphi = floor(crmax*pasDPHI)+1;
  const double convert = (double)(Ndphi-1)/(crmax+1./pasDPHI);


  int nbx = nrows / matcov_thread_x + (nrows%matcov_thread_x != 0);
  int nby = ncols / matcov_thread_y + (ncols%matcov_thread_y != 0);

  dim3 dimBlock(matcov_thread_x, matcov_thread_y);
  dim3 dimGrid(nbx, nby);
  const long Nsubap = tomo.Nsubap[Nw-1];
  
  matts_gpu_kernel<<<dimGrid, dimBlock>>>(data, nrows, ncols, xoffset, yoffset, lda, 
					   convert,tomo_gpu->X_d,tomo_gpu->Y_d,tomo_gpu->Nssp_d,
					   pasDPHI,tomo_gpu->tabDPHI_d,tomo_gpu->indexL0_d,tomo_gpu->cn2_d,
					   Ndphi,tomo.Nw,tomo.Nlayer,Nsubap,tomo.DiamTel);
  CudaCheckError();
}



__device__ double compute_element_noise(int ipos, int jpos, double convert, double *sspSizeL, long *Nssp, double *u, double *v, 
					double pasDPHI, double *tabDPHI, long *indexL0, double *cn2, int Ndphi, int Nw, int Nlayer, 
					int Nsubap, double *alphaX, double *alphaY, double lgs_cst, double noise_var, double spotWidth,
					double dH_lgs, double alt_lgs, int type_mat, int nlgs, double teldiam)
{
	/* *** Covariance matrix per-element generation ***
	*   Arguments
	*   =========
	*	ipos:		Integer: global x-coordinate of the element w.r.t. the entire matrix
	*	jpos:		Integer: global y-coordinate of the element w.r.t. the entire matrix
	*/
	
	// for now return a dummy value
  
  const double lambda2 = 0.00026942094446267851;
  //WFS m
  int m = ipos / (2 * Nsubap);
  if (type_mat == 3) m = Nw-1;
  //WFS n
  int n = jpos / (2 * Nsubap);
  if (type_mat == 2) n = Nw-1;
  //subap i
  int i = ipos % (2 * Nsubap); 
  //subap j
  int j = jpos % (2 * Nsubap);
  //xy i
  int xy_i;
  //xy j
  int xy_j;
  if (i>=Nsubap) {
    i-= Nsubap;
    xy_i = 1;
  } else xy_i = 0;
  if (j>=Nsubap) {
    j-= Nsubap;
    xy_j = 1;
  } else xy_j = 0;

  const double sspSizem = teldiam / Nssp[m];
  const double sspSizen = teldiam / Nssp[n];
  
  const double kk = lambda2 / (sspSizem * sspSizen);
    
  int type = xy_i * 2 + xy_j;

  //Layer l
  double covar = 0.0;
  #pragma unroll
  for (int l = 0; l < Nlayer; l++) 
  {
    const double sspSizeml = sspSizeL[m * Nlayer + l];
    const double sspSizenl = sspSizeL[n * Nlayer + l];
    //test if the altitude layers is not higher than the LGS altitude
    if ((sspSizeml > 0) && (sspSizenl > 0)) 
    {
      const int pos1 = m + i * Nw + l * Nw * Nsubap;
      const int pos2 = n + j * Nw + l * Nw * Nsubap;
      const double du = u[pos1] - u[pos2];	      
      const double dv =  v[pos1] - v[pos2];
      
      const double s1 = sspSizeml * 0.5;
      const double s2 = sspSizenl * 0.5;
      
      const double ac = s1 - s2;
      const double ad = s1 + s2;
      const double bc = -ad;   // initially -s1-s2;
      const double bd = -ac;   // initially -s1+s2;

      if (type == 0) covar += 0.5 * pasDPHI * cov_XX(du,dv,ac,ad,bc,bd,tabDPHI,indexL0[l],convert,Ndphi) * kk * cn2[l];
      else if (type == 3) covar += 0.5 * pasDPHI * cov_YY(du,dv,ac,ad,bc,bd,tabDPHI,indexL0[l],convert,Ndphi) * kk * cn2[l];
      else //if ((type == 1) || (type == 2)) 
      {
      	const double s0 = sqrt(s1 * s1 + s2 * s2); //half size of the subaperture equivalent to a convolution by s1 and s2
      	const double dd = (s1 > s2) ? 1. - s2 / s1 : 1. - s1 / s2; // Nono's style ....
      	covar += 0.25 * pasDPHI * cov_XY(du,dv,s0,tabDPHI,indexL0[l],convert,Ndphi) * kk * cn2[l] * (1. - dd * dd);
      }
    }
  }
  // adding noise
  if (m == n) {
    if (m < nlgs) {
      if (i == j) {
	// lgs case
	const int pos1 = m + i * Nw;
	double x = u[pos1];	      
	double y = v[pos1];
	const double xwfs = alphaX[m] * 206265;	      
	const double ywfs = alphaY[m] * 206265;
	double lltx = 0;	      
	double llty = 0;
	const double lltnorm = sqrtf(xwfs*xwfs + ywfs*ywfs);
	if (lltnorm != 0) {
	  lltx = xwfs / lltnorm * teldiam / 2.0;
	  llty = ywfs / lltnorm * teldiam / 2.0;
	}
	x -= lltx;
	y -= llty;
        x  = 206265. * dH_lgs * x / alt_lgs / alt_lgs;   // extension at Fwhm, in arcsec
        y  = 206265. * dH_lgs * y / alt_lgs / alt_lgs;   // extension at Fwhm, in arcsec
        const double lgsExt = sqrtf(x * x + y * y);   // lengh of the extension
        const double lgsTheta = x != 0 ? atanf( y / x) : 0.0;   // angle of extension
        const double totalExt = sqrtf( lgsExt *  lgsExt + spotWidth * spotWidth); 
	// lengh of the extension including seeing, laser size, ...
	const double ratio = totalExt / spotWidth;
        const double noiseLongAxis = noise_var * ratio * ratio;
	if (type == 0) covar += noiseLongAxis * cosf(lgsTheta) * cosf(lgsTheta) + 
			 noise_var * sinf(lgsTheta) * sinf(lgsTheta);
	else if (type == 3) covar += noiseLongAxis * sinf(lgsTheta) * sinf(lgsTheta) + 
			      noise_var * cosf(lgsTheta) * cosf(lgsTheta);
	else covar += (noiseLongAxis-noise_var) * sinf(lgsTheta) * cosf(lgsTheta);
      }
      if ((type == 0) || (type == 3))
	covar += lgs_cst;
    } else {
    // ngs case
      if (i==j) {
	if ((type == 0) || (type == 3)) {
	  covar += noise_var;
	}
      }
    }
  }

  return (double)covar; 
}

__global__ void matcovnoise_gpu_kernel(double* data, int nrows, int ncols, int xoffset, int yoffset, int lda,
				       double convert, double *sspSizeL, long *Nssp, double *u, double *v, 
					double pasDPHI, double *tabDPHI, long *indexL0, double *cn2, int Ndphi, int Nw, int Nlayer, 
					int Nsubap, double *alphaX, double *alphaY, double lgs_cst, double noise_var, double spotWidth,
					double dH_lgs, double alt_lgs, int type_mat, int nlgs, double teldiam)
{
  /* *** covariance matrix generation kernel ***
   *	The kernel generates the element values in a given matrix/submatrix
   *   The generation function can be any function, as long as each element 
   *   can be computed both individually and independently
   *
   *	see argument description in the kernel driver
   */
	
  // local thread coordinates w.r.t. thread block
  const int tx_ = threadIdx.x;
  const int ty_ = threadIdx.y;
	
  // local thread block coordinates w.r.t. kernel grid
  const int bx_ = blockIdx.x;
  const int by_ = blockIdx.y;
	
  // local coordinates of the element w.r.t. submatrix
  int lx = bx_ * blockDim.x + tx_;
  int ly = by_ * blockDim.y + ty_;
	
  // global coordinates of the elemnt w.r.t. the entire matrix
  int gx = lx + xoffset;
  int gy = ly + yoffset;
	
  // out-of-bound threads should terminate
  if( (lx >= nrows) || (ly >= ncols) ) return;
	
  // Advance the data pointer accordingly
  data += ly * lda + lx;
	
  if ((type_mat == 3) || (gx <= gy)) {
    // call the generation function
    data[0] = compute_element_noise(gx, gy, convert, sspSizeL, Nssp, u, v, pasDPHI, tabDPHI, indexL0, cn2, Ndphi, Nw, Nlayer, 
					Nsubap, alphaX, alphaY, lgs_cst, noise_var, spotWidth, dH_lgs, alt_lgs, type_mat, nlgs, teldiam);
    //printf("gx = %d, gy = %d ----- %.2f \n", gx, gy, data[0]);
  } 
}

void matcov_gpu4(double* data, int nrows, int ncols, int xoffset, int yoffset, int lda, struct tomo_struct tomo, 
		struct tomo_gpu_struct *tomo_gpu)
{
	/* *** matcov gpu kernel driver ***
	*  Arguments
	*  ==========
	*  data		double pointer: A pointer to the matrix/submatrix to be generated. It  
	*  			should always point to the first element in a matrix/submatrix
	*
	*  nrows	integer: The number of rows of the matrix/submatrix to be generated	
	*
	*  ncols	integer: The number of columns of the matrix/submatrix to be generated
	*
	*  xoffset	integer: The x-offset of the submatrix, must be zero if the entire matrix
	*			is generated. Its the x-coordinate of the first element in the matrix/submatrix
	*
	*  yoffset  integer: The y-offset of the submatrix, must be zero if the entire matrix
	*			is generated. Its the y-coordinate of the first element in the matrix/submatrix
	*
	*  lda		integer: The leading dimension of the matrix/submatrix
	*/
	
  // %%%%%%% Pre-computation of DPHI %%%%%%%%%%
  //Computes an array of DPHI (tabDPHI) for an array of subaperture distance rr for each DIFFERENT L0
  const double crmax = tomo.rmax;
  const double pasDPHI = 1./tomo.pasDPHI; //inverse du pas de rr
  const long Ndphi = floor(crmax*pasDPHI)+1;
  const double convert = (double)(Ndphi-1)/(crmax+1./pasDPHI);

  int type_mat = tomo.part;

  int nbx = nrows / matcov_thread_x + (nrows%matcov_thread_x != 0);
  int nby = ncols / matcov_thread_y + (ncols%matcov_thread_y != 0);

  dim3 dimBlock(matcov_thread_x, matcov_thread_y);
  dim3 dimGrid(nbx, nby);
  const long Nsubap = tomo.Nsubap[0];
  
  matcovnoise_gpu_kernel<<<dimGrid, dimBlock>>>(data, nrows, ncols, xoffset, yoffset, lda, convert, tomo_gpu->sspSizeL_d, 
						tomo_gpu->Nssp_d, tomo_gpu->u_d, tomo_gpu->v_d, pasDPHI, tomo_gpu->tabDPHI_d, 
						tomo_gpu->indexL0_d, tomo_gpu->cn2_d, Ndphi, tomo.Nw, tomo.Nlayer, 
						Nsubap, tomo_gpu->alphaX_d, tomo_gpu->alphaY_d, tomo.lgs_cst, tomo.noise_var, 
						tomo.spot_width, tomo.lgs_depth, tomo.lgs_alt, type_mat, tomo.nlgs, tomo.DiamTel);

  if (type_mat == 1)
    matcov_gpu_kernel_copy<<<dimGrid, dimBlock>>>(data, nrows, ncols, xoffset, yoffset, lda);
}
